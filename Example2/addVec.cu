#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

__global__
void add(int n, float *x, float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n){
    z[i] = x[i] + y[i];
  }
}

double cpuSecond(){
  struct timeval tp;
  gettimeofday(&tp, NULL);
  double timeRet=(double)tp.tv_sec + (double)tp.tv_usec*1e-6;
  return timeRet;


}


int main(int argc, char **argv)
{
  int N = 1 << 24;
  
  float*x = (float*)malloc(N*sizeof(float));
  float*y = (float*)malloc(N*sizeof(float));
  float*z = (float*)malloc(N*sizeof(float));
  float *zcpu = (float*)malloc(N*sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = i;
    y[i] = 42-i;
  }

#if 0
  double cpuStart=cpuSecond();
  for (int i = 0; i < N; i++) {
    zcpu[i] = x[i] + y[i];
  }
 double cpuEnd=cpuSecond();
 double cpuElapse = cpuEnd - cpuStart;
 printf("Time take on CPU %f s\n", cpuElapse);
#endif

  // note the reference of a *pointer*!
  float *x_c, *y_c, *z_c;
  hipMalloc(&x_c, N*sizeof(float)); 
  hipMalloc(&y_c, N*sizeof(float));
  hipMalloc(&z_c, N*sizeof(float));

  // copy host memory over to GPU 
  hipMemcpy(x_c, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_c, y, N*sizeof(float), hipMemcpyHostToDevice);

  // add vectors together
  int Nthreads = atoi(argv[1]);
  dim3 threadsPerBlock(Nthreads,1,1);
  dim3 blocks((N+Nthreads-1)/Nthreads,1,1);
  double iStart=cpuSecond();
  add <<< blocks, threadsPerBlock >>> (N, x_c, y_c, z_c);
  // copy result z_c back to CPU (in z)
  hipMemcpy(z, z_c, N*sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  double iStop=cpuSecond();
  // check result
  double elapsed=iStop - iStart;
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(z[i]-42.f));

  printf("Max error: %f\n", maxError);
  printf("AddVec Kernel on GPU with <<<%d, %d>>> and Elapsed time is %f sec \n", blocks.x, threadsPerBlock.x, elapsed);

  // free memory on both CPU and GPU
  hipFree(x_c);
  hipFree(y_c);
  free(x);
  free(y);
}
